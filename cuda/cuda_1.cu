#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
using namespace std;
const int X = 1024;
size_t threadsPerBlock;
size_t numberOfBlocks;
//  Initial array
void A_reset(float *A)
{
    for (int i = 0; i < X; i++)
    {
        for (int j = 0; j < i; j++)
            A[i*X+j] = 0;
        A[i*X+i] = 1.0;
        for (int j = i + 1; j < X; j++)
            A[i*X+j] = rand();
    }
    for (int k = 0; k < X; k++)
        for (int i = k + 1; i < X; i++)
            for (int j = 0; j < X; j++)
                A[i*X+j] += A[k*X+j];
}

// serial
void normal(int n, float **A)
{
    for (int k = 0; k < n; k++)
    {
        for (int j = k + 1; j < n; j++)
        {
            A[k*n+j] /= A[k*n+k];
        }
        A[k*n+k] = 1.0;
        for (int i = k + 1; i < n; i++)
        {
            for (int j = k + 1; j < n; j++)
            {
                A[i*n+j] -= A[i*n+k] * A[k*n+j];
            }
            A[i][k] = 0;
        }
    }
}
__global__ void division_kernel(float **A, int k, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index+k+1; i < n ; i += stride)
    {
        float element = A[k][k];
        float temp = A[k][i];
        //请同学们思考，如果分配的总线程数小于 N 应该怎么办？
        A[k][i] = (float)temp / element;
    }
    return;
}

__global__ void eliminate_kernel(float **A, int k, int N)
{
    int tx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tx == 0)
        A[k][k] = 1;    //对角线元素设为 1
    int row = k + 1 + blockIdx.x; //每个块负责一行
    while (row < N)
    {
        int tid = threadIdx.x;
        while (k + 1 + tid < N)
        {
            int col = k + 1 + tid;
            float temp_1 = A[row][col];
            float temp_2 = A[row][k];
            float temp_3 = A[k][col];
            A[row][col] = temp_1 - temp_2 * temp_3;
            tid = tid + blockDim.x;
        }
        __syncthreads(); //块内同步
        if (threadIdx.x == 0)
        {
            A[row][k] = 0;
        }
        row += gridDim.x;
    }
    return;
}

//并行算法
void sp(int n, float **A)
{
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    threadsPerBlock = 32;
    numberOfBlocks = 32 * numberOfSMs;

    // size_t size = n * n * sizeof(float);
    // float **A_d;
    // hipMalloc((void **)&A_d, size);
    // hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);

    for (int k = 0; k < n; k++)
    {
        division_kernel<<<numberOfBlocks, threadsPerBlock>>>(A, k, n); //负责除法任务的核函数
        hipDeviceSynchronize();                                 // CPU 与 GPU 之间的同步函数
        // ret = hipGetLastError();
        // if (ret != hipSuccess)
        // {
        //     printf("division_kernel failed, %s\n", hipGetErrorString(ret));
        // }
        eliminate_kernel<<<numberOfBlocks, threadsPerBlock>>>(A, k, n); //负责消去任务的核函数
        hipDeviceSynchronize();
        // ret = hipGetLastError();
        // if (ret != hipSuccess)
        // {
        //     printf("eliminate_kernel failed, %s\n", hipGetErrorString(ret));
        // }
    }

    //hipMemcpy(A, A_d, size, hipMemcpyDeviceToHost);

    //hipFree(A);

}
int main()
{
    
    size_t size = X * X * sizeof(float);
    float *A;
    hipMallocManaged(&A, size);
    
    int step = 64;
    clock_t start ,finish_1,finish_2;
    for (int i = step; i <= X; i += step)
    {
        //串行
        A_reset(A);
        start = clock();
        normal(i, A);
        finish_1 = clock();
        float time_1 = ( finish_1 - start)/float (CLOCKS_PER_SEC);
        //并行
        A_reset(A);
        start = clock();
        sp(i, A);
        finish_2 = clock();
        float time_2 = ( finish_2 - start)/float (CLOCKS_PER_SEC);
        cout<<fixed << setprecision(6);
        cout<< time_1 << "    " << time_2 << endl;
        
    }
    hipFree(A);
    cout << "hello" << endl;
    return 0;
}